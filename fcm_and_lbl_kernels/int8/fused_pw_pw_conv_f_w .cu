#include "hip/hip_runtime.h"
#include "../../headers/conv_kernels.h"

#if COMPILE_FUSED && (FUSION_MODE == ALL_MODES || FUSION_MODE == FUSED_F_W) && DATA_TYPE == INT8_DTYPE

using namespace std;
namespace cg = cooperative_groups;

void inline __device__ fill_dw_weights_tile(weights_dt *dw_weights,
                                            int8_t *weights_tile,
                                            const int dw_filter_area,
                                            const int layer_weights_offset,
                                            const int compact_layer_depth,
                                            const int start_filling_depth,
                                            const int offset_in_tile_depth)
{
    for (int i = 0; i < dw_filter_area; i++)
    {
        weights_dt current_weight = dw_weights[layer_weights_offset + i * compact_layer_depth + start_filling_depth];
        for (int d = 0; d < PACKED_ITEMS; d++)
        {
            weights_tile[(offset_in_tile_depth + d) * dw_filter_area + i] = EXTRACT_8_32(current_weight, d);
        }
    }
}

void inline __device__ fill_dw_scales_tile(scales_dt *fused_scales,
                                           biases_dt *fused_zps,
                                           scales_dt *fused_scales_tile,
                                           biases_dt *fused_zps_tile,
                                           const int start_filling_depth,
                                           const int fused_params_offset,
                                           const int offset_in_tile_depth)
{
    for (int f = 0; f < PACKED_ITEMS; f++)
    {
        fused_scales_tile[offset_in_tile_depth + f] = fused_scales[fused_params_offset + start_filling_depth + f];
        fused_zps_tile[offset_in_tile_depth + f] = fused_zps[fused_params_offset + start_filling_depth + f];
    }
}

__global__ void pw_pw_conv_f_w(fms_dt *ifms, fms_dt *ofms, weights_dt *pw_weights,
                               fused_scales_dt *fused_scales,
                               biases_dt *fused_zps,
                               const int pw_1_compact_layer_depth,
                               const int pw_1_compact_layer_num_filters,
                               const int pw_1_ifm_width,
                               const int pw_1_compact_ifm_width_depth,
                               const int pw_1_ofm_height,
                               const int pw_1_ofm_width,
                               const int pw_1_compact_ofm_width_depth,
                               const int pw_1_depth_to_parallelism_f_ratio,
                               const int pw_1_layer_weights_offset,
                               const int pw_1_layer_fused_params_offset,
                               const fms_dt pw_1_ofms_zp,
                               const scales_dt pw_1_relu_threshold,
                               const int pw_1_layer_activation,
                               const int pw_2_compact_layer_depth,
                               const int pw_2_compact_layer_num_filters,
                               const int pw_2_ifm_width,
                               const int pw_2_compact_ifm_width_depth,
                               const int pw_2_ofm_height,
                               const int pw_2_ofm_width,
                               const int pw_2_compact_ofm_width_depth,
                               const int pw_2_filters_to_parallelism_f_ratio,
                               const int pw_2_layer_weights_offset,
                               const int pw_2_layer_fused_params_offset,
                               const fms_dt pw_2_ofms_zp,
                               const scales_dt pw_2_relu_threshold,
                               const int parallel_w)
{

    const int thread_w = threadIdx.x;
    const int thread_f = threadIdx.z;

    const int pw_1_filter_index = thread_f * PACKED_ITEMS;

    const int block_w = blockIdx.x;
    const int abs_h_index = blockIdx.y;

    __shared__ fms_dt ofms_ifms_tile[PW_PW_MAX_FMS_BUFFER_SZ];
    //__shared__ fms_dt pw_2_weights[PW_PW_MAX_WEIGHTS_BUFFER_SZ];

    scales_dt scale0, scale1, scale2, scale3;

    biases_dt fused_zp0, fused_zp1, fused_zp2, fused_zp3;

    const int abs_w_index = block_w * parallel_w + thread_w;

    scale0 = fused_scales[pw_1_layer_fused_params_offset + pw_1_filter_index];
    scale1 = fused_scales[pw_1_layer_fused_params_offset + pw_1_filter_index + 1];
    scale2 = fused_scales[pw_1_layer_fused_params_offset + pw_1_filter_index + 2];
    scale3 = fused_scales[pw_1_layer_fused_params_offset + pw_1_filter_index + 3];

    fused_zp0 = fused_zps[pw_1_layer_fused_params_offset + pw_1_filter_index];
    fused_zp1 = fused_zps[pw_1_layer_fused_params_offset + pw_1_filter_index + 1];
    fused_zp2 = fused_zps[pw_1_layer_fused_params_offset + pw_1_filter_index + 2];
    fused_zp3 = fused_zps[pw_1_layer_fused_params_offset + pw_1_filter_index + 3];

    const int base_index_pw_1_weights = pw_1_layer_weights_offset + pw_1_filter_index * pw_1_compact_layer_depth;

    const int base_index_in_ifms = abs_h_index * pw_1_ifm_width + abs_w_index;
    
    if (abs_w_index < pw_1_ofm_width)
    {
        pss_dt sum0 = 0, sum1 = 0, sum2 = 0, sum3 = 0;
        int a = 0;
        for (int d = 0; d < pw_1_compact_layer_depth; d++)
        {
            fms_dt fms_val = ifms[base_index_in_ifms + d * pw_1_ifm_width * pw_1_ifm_width];
            // ifms_tile[d * parallel_w + thread_w];

            sum0 += __dp4a(fms_val, pw_weights[base_index_pw_1_weights + d], a);
            sum1 += __dp4a(fms_val, pw_weights[base_index_pw_1_weights + pw_1_compact_layer_depth + d], a);
            sum2 += __dp4a(fms_val, pw_weights[base_index_pw_1_weights + 2 * pw_1_compact_layer_depth + d], a);
            sum3 += __dp4a(fms_val, pw_weights[base_index_pw_1_weights + 3 * pw_1_compact_layer_depth + d], a);
        }
        if (pw_1_layer_activation == 0)
        {
            ofms_ifms_tile[thread_f * parallel_w + thread_w] =
                PACK_32_8s(quant_no_activation(sum0, scale0, fused_zp0, pw_1_ofms_zp),
                           quant_no_activation(sum1, scale1, fused_zp1, pw_1_ofms_zp),
                           quant_no_activation(sum2, scale2, fused_zp2, pw_1_ofms_zp),
                           quant_no_activation(sum3, scale3, fused_zp3, pw_1_ofms_zp));
        }
        else if (pw_1_layer_activation == RELU6)
        {
            ofms_ifms_tile[thread_f * parallel_w + thread_w] =
                PACK_32_8s(quant_relu6(sum0, scale0, fused_zp0, pw_1_ofms_zp, pw_1_relu_threshold),
                           quant_relu6(sum1, scale1, fused_zp1, pw_1_ofms_zp, pw_1_relu_threshold),
                           quant_relu6(sum2, scale2, fused_zp2, pw_1_ofms_zp, pw_1_relu_threshold),
                           quant_relu6(sum3, scale3, fused_zp3, pw_1_ofms_zp, pw_1_relu_threshold));
        }
    }
    //}

    __syncthreads();

    if (abs_w_index < pw_2_ofm_width)
    {
        // const int abs_w_index = block_w * parallel_w + (thread_w - parallel_w);

        for (int o_f = 0; o_f < pw_2_filters_to_parallelism_f_ratio; o_f++)
        {
            const int pw_2_compact_filter_index = thread_f * pw_2_filters_to_parallelism_f_ratio + o_f;
            const int pw_2_filter_index = pw_2_compact_filter_index * PACKED_ITEMS;

            if (pw_2_compact_filter_index < pw_2_compact_layer_num_filters)
            {
                const int f_offset = pw_2_filter_index * pw_2_compact_layer_depth;
                const int base_index_pw_2_weights = pw_2_layer_weights_offset + f_offset;

                scale0 = fused_scales[pw_2_layer_fused_params_offset + pw_2_filter_index];
                scale1 = fused_scales[pw_2_layer_fused_params_offset + pw_2_filter_index + 1];
                scale2 = fused_scales[pw_2_layer_fused_params_offset + pw_2_filter_index + 2];
                scale3 = fused_scales[pw_2_layer_fused_params_offset + pw_2_filter_index + 3];

                fused_zp0 = fused_zps[pw_2_layer_fused_params_offset + pw_2_filter_index];
                fused_zp1 = fused_zps[pw_2_layer_fused_params_offset + pw_2_filter_index + 1];
                fused_zp2 = fused_zps[pw_2_layer_fused_params_offset + pw_2_filter_index + 2];
                fused_zp3 = fused_zps[pw_2_layer_fused_params_offset + pw_2_filter_index + 3];

                if (abs_w_index < pw_2_ofm_width)
                {
                    const int base_index_in_ofms = abs_h_index * pw_1_ofm_width +
                                   pw_2_compact_filter_index * pw_1_ofm_height * pw_1_ofm_width + abs_w_index;

                    pss_dt sum0 = 0, sum1 = 0, sum2 = 0, sum3 = 0;
                    int a = 0;
                    for (int d = 0; d < pw_2_compact_layer_depth; d++)
                    {
                        fms_dt fms_val = ofms_ifms_tile[d * parallel_w + thread_w];

                        sum0 += __dp4a(fms_val, pw_weights[base_index_pw_2_weights + d], a); 
                        sum1 += __dp4a(fms_val, pw_weights[base_index_pw_2_weights + pw_2_compact_layer_depth + d], a);
                        sum2 += __dp4a(fms_val, pw_weights[base_index_pw_2_weights + 2 * pw_2_compact_layer_depth + d], a);
                        sum3 += __dp4a(fms_val, pw_weights[base_index_pw_2_weights + 3 * pw_2_compact_layer_depth + d], a);
                    }
                    ofms[base_index_in_ofms] = PACK_32_8s(quant_relu6(sum0, scale0, fused_zp0, pw_2_ofms_zp, pw_2_relu_threshold),
                                                          quant_relu6(sum1, scale1, fused_zp1, pw_2_ofms_zp, pw_2_relu_threshold),
                                                          quant_relu6(sum2, scale2, fused_zp2, pw_2_ofms_zp, pw_2_relu_threshold),
                                                          quant_relu6(sum3, scale3, fused_zp3, pw_2_ofms_zp, pw_2_relu_threshold));
                }
            }
        }
    }
}

void fused_pw_pw_convolutionGPU_chw(fms_dt *ifms, fms_dt *ofms,
                                weights_dt *pw_weights,
                                fused_scales_dt *fused_scales,
                                biases_dt *fused_zps,
                                layer_specs pw_1_l_specs,
                                layer_specs pw_2_l_specs,
                                int *fused_params_offsets,
                                const int iteration,
                                int *layers_parallelism_w,
                                float &exec_time)
{

    const int pw_1_compact_layer_depth = (pw_1_l_specs.layer_depth / PACKED_ITEMS);
    const int pw_1_compact_layer_num_filters = (pw_1_l_specs.layer_num_fils / PACKED_ITEMS);

    int parallel_w = layers_parallelism_w[pw_1_l_specs.layer_index];

    dim3 threads(parallel_w, 1, pw_1_compact_layer_num_filters);
    dim3 blocks((pw_1_l_specs.layer_ifm_width + parallel_w - 1) / parallel_w,
                pw_1_l_specs.layer_ifm_height, 1);

#if TIME_LAYER_BY_LAYER
    float elapsed_time;
    hipEvent_t start_event, stop_event;
    hipError_t err = hipSuccess;

    err = (hipEventCreate(&start_event));
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventCreate start_event %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = (hipEventCreate(&stop_event));
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventCreate stop_event %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    if (iteration == 0)
    {
        printf("%d, %d (FUSED_PWPW):\n", pw_1_l_specs.layer_index, pw_2_l_specs.layer_index);
    }

    err = hipEventRecord(start_event, 0);
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventRecord start_event %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
#endif

    const int pw_1_layer_ifm_width = pw_1_l_specs.layer_ifm_width;
    const int pw_1_layer_ofm_width = pw_1_l_specs.layer_ofm_width;
    const int pw_1_compact_ofms_width_depth = pw_1_compact_layer_num_filters * pw_1_layer_ofm_width;
    const int pw_2_layer_depth = pw_2_l_specs.layer_depth;
    const int pw_2_layer_ifm_width = pw_2_l_specs.layer_ifm_width;
    const int pw_2_layer_ofm_width = pw_2_l_specs.layer_ofm_width;
    const int pw_2_compact_layer_num_filters = pw_2_l_specs.layer_num_fils / PACKED_ITEMS;
    const int pw_2_compact_ofms_width_depth = pw_2_compact_layer_num_filters * pw_2_layer_ofm_width;
    const int pw_2_compact_layer_depth = pw_2_layer_depth / PACKED_ITEMS;

    int pw_1_depth_to_parallelism_f_ratio = pw_1_compact_layer_depth / pw_1_compact_layer_num_filters;
    if (pw_1_depth_to_parallelism_f_ratio < 1)
    {
        pw_1_depth_to_parallelism_f_ratio = 1;
    }
    int pw_2_filters_to_parallelism_f_ratio = pw_2_compact_layer_num_filters / pw_1_compact_layer_num_filters;
    if (pw_2_filters_to_parallelism_f_ratio < 1)
    {
        pw_2_filters_to_parallelism_f_ratio = 1;
    }

    pw_pw_conv_f_w<<<blocks, threads>>>(ifms, ofms, pw_weights,
                                        fused_scales,
                                        fused_zps,
                                        pw_1_compact_layer_depth,
                                        pw_1_compact_layer_num_filters,
                                        pw_1_layer_ifm_width,
                                        pw_1_compact_layer_depth * pw_1_layer_ifm_width,
                                        pw_1_l_specs.layer_ofm_height,
                                        pw_1_l_specs.layer_ofm_width,
                                        pw_1_compact_ofms_width_depth,
                                        pw_1_depth_to_parallelism_f_ratio,
                                        pw_1_l_specs.layer_weights_offset / PACKED_ITEMS,
                                        fused_params_offsets[pw_1_l_specs.layer_index],
                                        pw_1_l_specs.layer_ofms_zero_point,
                                        pw_1_l_specs.relu_threshold,
                                        pw_1_l_specs.layer_activation,
                                        pw_2_compact_layer_depth,
                                        pw_2_compact_layer_num_filters,
                                        pw_2_layer_ifm_width,
                                        pw_2_compact_layer_depth * pw_2_layer_ifm_width,
                                        pw_2_l_specs.layer_ofm_height,
                                        pw_2_l_specs.layer_ofm_width,
                                        pw_2_compact_ofms_width_depth,
                                        pw_2_filters_to_parallelism_f_ratio,
                                        pw_2_l_specs.layer_weights_offset / PACKED_ITEMS,
                                        fused_params_offsets[pw_2_l_specs.layer_index],
                                        pw_2_l_specs.layer_ofms_zero_point,
                                        pw_2_l_specs.relu_threshold,
                                        parallel_w);

#if TIME_LAYER_BY_LAYER
    err = (hipEventRecord(stop_event, 0));
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventRecord stop_event %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = (hipEventSynchronize(stop_event));
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventSynchronize %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = (hipEventElapsedTime(&elapsed_time, start_event, stop_event));
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventElapsedTime %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // printf("Measured time for sample = %.3fms\n", elapsed_time);
    if (iteration >= WARMUP_ITERATIONS)
    {
        exec_time += elapsed_time;
    }
#endif

    hipError_t kernel_error = hipGetLastError();
    if (kernel_error != hipSuccess)
    {
        cout << "the error of code: " << kernel_error << " has happened\n";
    }
}

#endif