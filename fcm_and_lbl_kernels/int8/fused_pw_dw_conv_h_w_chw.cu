#include "hip/hip_runtime.h"
#include "../../headers/conv_kernels.h"

#if COMPILE_FUSED && (FUSION_MODE == ALL_MODES || FUSION_MODE == FUSED_H_W) && DATA_LAYOUT == CHW && DATA_TYPE == INT8_DTYPE

using namespace std;
namespace cg = cooperative_groups;


__global__ void pw_dw3x3_conv_h_w(fms_dt *ifms, fms_dt *ofms, weights_dt *pw_weights,
                                  fused_scales_dt *fused_scales,
                                  biases_dt *fused_zps,
                                  const int compact_layer_depth,
                                  const int pw_num_filters,
                                  const int pw_ifm_width,
                                  const int pw_ofm_height,
                                  const int pw_ofm_width,
                                  const int pw_layer_weights_offset,
                                  const int pw_layer_fused_params_offset,
                                  const fms_dt pw_ofms_zp,
                                  const scales_dt pw_relu_threshold,
                                  weights_dt *dw_weights,
                                  const int dw_ifm_depth,
                                  const int dw_ifm_height,
                                  const int dw_ifm_width,
                                  const int dw_ofm_height,
                                  const int dw_ofm_width,
                                  const int strides,
                                  const int padding_top,
                                  const int padding_bottom,
                                  const int padding_left,
                                  const int padding_right,
                                  const int padded_tile_width,
                                  const int dw_layer_weights_offset,
                                  const int dw_layer_fused_params_offset,
                                  const fms_dt dw_ifms_zp,
                                  const fms_dt dw_ofms_zp,
                                  const fms_dt packed_ifm_zp,
                                  const scales_dt dw_relu_threshold,
                                  const int parallel_h,
                                  const int parallel_w,
                                  const int compact_layer_depth_to_parallel_hw_ratio)
{

    const int thread_w = threadIdx.x;
    const int thread_h = threadIdx.y;
    const int thread_f = threadIdx.z;

    const int block_w = blockIdx.x;
    const int block_h = blockIdx.y;
    const int block_f = blockIdx.z;

    const int abs_h = block_h * parallel_h + thread_h;
    const int parallel_hw = parallel_h * parallel_w;
    const int thread_hw = thread_h * parallel_h + thread_w;
    const int abs_dw_f_compact = block_f + thread_f;
    const int abs_dw_f = abs_dw_f_compact * PACKED_ITEMS;

    const int base_index_pw_weights = pw_layer_weights_offset + block_f * PACKED_ITEMS * compact_layer_depth;

    __shared__ weights_dt weights_tile[MAX_PW_COMPACT_DEPTH_FUSED * PACKED_ITEMS];
    __shared__ fms_dt ofms_ifms_tile[TILE_H_H_W * TILE_W_H_W];

    __shared__ weights_dt dw_filter_weights[FILTER_3x3_AREA * PACKED_ITEMS];

    for (int i = 0; i < compact_layer_depth_to_parallel_hw_ratio; i++)
    {
        const int iter_offset = i * parallel_hw + thread_hw;
        if (iter_offset < compact_layer_depth) // TODO
        {
            weights_tile[iter_offset * PACKED_ITEMS] = pw_weights[base_index_pw_weights + iter_offset];
            weights_tile[iter_offset * PACKED_ITEMS + 1] = pw_weights[base_index_pw_weights + compact_layer_depth + iter_offset];
            weights_tile[iter_offset * PACKED_ITEMS + 2] = pw_weights[base_index_pw_weights + 2 * compact_layer_depth + iter_offset];
            weights_tile[iter_offset * PACKED_ITEMS + 3] = pw_weights[base_index_pw_weights + 3 * compact_layer_depth + iter_offset];
        }
    }
    __syncthreads();

    if (abs_h < pw_ofm_height)
    {
        const int abs_w_write = block_w * parallel_w + thread_w;
        if (abs_w_write < pw_ofm_width)
        {
            if (thread_h == 0 && thread_w == 0)
            {
                for (int c_h = 0; c_h < FILTER_3x3_DIM; c_h++)
                {
                    weights_dt weight_val0 =
                        dw_weights[dw_layer_weights_offset + (c_h * FILTER_3x3_DIM) + abs_dw_f_compact * FILTER_3x3_PADDED_AREA];
                    weights_dt weight_val1 =
                        dw_weights[dw_layer_weights_offset + (c_h * FILTER_3x3_DIM + 1) + abs_dw_f_compact * FILTER_3x3_PADDED_AREA];
                    weights_dt weight_val2 =
                        dw_weights[dw_layer_weights_offset + (c_h * FILTER_3x3_DIM + 2) + abs_dw_f_compact * FILTER_3x3_PADDED_AREA];
                    for (int f = 0; f < PACKED_ITEMS; f++)
                    {
                        dw_filter_weights[f * FILTER_3x3_DIM + c_h] = PACK_32_8s(EXTRACT_8_32(weight_val0, f),
                                                                                 EXTRACT_8_32(weight_val1, f),
                                                                                 EXTRACT_8_32(weight_val2, f),
                                                                                 0);
                    }
                }
            }

            const int offet_in_tile = thread_h * parallel_w + thread_w;
            const int base_index_pw_scales = block_f * PACKED_ITEMS;

            scales_dt scale0 = fused_scales[pw_layer_fused_params_offset + base_index_pw_scales];
            scales_dt scale1 = fused_scales[pw_layer_fused_params_offset + base_index_pw_scales + 1];
            scales_dt scale2 = fused_scales[pw_layer_fused_params_offset + base_index_pw_scales + 2];
            scales_dt scale3 = fused_scales[pw_layer_fused_params_offset + base_index_pw_scales + 3];

            biases_dt fused_zp0 = fused_zps[pw_layer_fused_params_offset + base_index_pw_scales];
            biases_dt fused_zp1 = fused_zps[pw_layer_fused_params_offset + base_index_pw_scales + 1];
            biases_dt fused_zp2 = fused_zps[pw_layer_fused_params_offset + base_index_pw_scales + 2];
            biases_dt fused_zp3 = fused_zps[pw_layer_fused_params_offset + base_index_pw_scales + 3];

            pss_dt sum0 = 0, sum1 = 0, sum2 = 0, sum3 = 0;
            int base_index_in_ifms = abs_h * pw_ifm_width + abs_w_write;
            const int pw_ifms_hw = pw_ifm_width * pw_ifm_width; // TODO
            for (int d = 0; d < compact_layer_depth; d++)
            {
                const int d_offset = d * PACKED_ITEMS;

                fms_dt ifms_val = ifms[base_index_in_ifms + d * pw_ifms_hw];

                sum0 += __dp4a(ifms_val, weights_tile[d_offset], 0);
                sum1 += __dp4a(ifms_val, weights_tile[d_offset + 1], 0);
                sum2 += __dp4a(ifms_val, weights_tile[d_offset + 2], 0);
                sum3 += __dp4a(ifms_val, weights_tile[d_offset + 3], 0);

                // if (base_index_in_ofms + abs_w_write < 10)
                // {
                //     printf("%d * %d\n ", EXTRACT_8_32(pw_weights[base_index_pw_weights + d], 0), EXTRACT_8_32(fms_val, 0));
                //     printf("%d * %d\n", EXTRACT_8_32(pw_weights[base_index_pw_weights + d], 1), EXTRACT_8_32(fms_val, 1));
                //     printf("%d * %d\n", EXTRACT_8_32(pw_weights[base_index_pw_weights + d], 2), EXTRACT_8_32(fms_val, 2));
                //     printf("%d * %d\n", EXTRACT_8_32(pw_weights[base_index_pw_weights + d], 3), EXTRACT_8_32(fms_val, 3));
                // }
            }

            // if (base_index_in_ofms + abs_w_write < 10)
            // {
            //     printf("%d \n", quant_relu6(sum0, scale0, fused_zp0, pw_ofms_zp, pw_relu_threshold));
            // }
            ofms_ifms_tile[offet_in_tile] = PACK_32_8s(quant_relu6(sum0, scale0, fused_zp0, pw_ofms_zp, pw_relu_threshold),
                                                       quant_relu6(sum1, scale1, fused_zp1, pw_ofms_zp, pw_relu_threshold),
                                                       quant_relu6(sum2, scale2, fused_zp2, pw_ofms_zp, pw_relu_threshold),
                                                       quant_relu6(sum3, scale3, fused_zp3, pw_ofms_zp, pw_relu_threshold));

            //**********************************************************
            __syncthreads();

            scale0 = fused_scales[dw_layer_fused_params_offset + abs_dw_f];
            scale1 = fused_scales[dw_layer_fused_params_offset + abs_dw_f + 1];
            scale2 = fused_scales[dw_layer_fused_params_offset + abs_dw_f + 2];
            scale3 = fused_scales[dw_layer_fused_params_offset + abs_dw_f + 3];

            fused_zp0 = fused_zps[dw_layer_fused_params_offset + abs_dw_f];
            fused_zp1 = fused_zps[dw_layer_fused_params_offset + abs_dw_f + 1];
            fused_zp2 = fused_zps[dw_layer_fused_params_offset + abs_dw_f + 2];
            fused_zp3 = fused_zps[dw_layer_fused_params_offset + abs_dw_f + 3];
            // for (int h = 0; h < rows_per_thread; h++)
            {
                const int row_index_in_tile = thread_h * strides - padding_top;

                const int base_index_in_ofms = abs_dw_f_compact * dw_ofm_height * dw_ofm_width +
                                               (block_h * parallel_h / strides + thread_h) *
                                                   dw_ofm_width + abs_w_write;
                if (thread_h < parallel_h / strides)
                {
                    sum0 = 0, sum1 = 0, sum2 = 0, sum3 = 0;
                    const int abs_w_read = thread_w * strides - padding_left;

                    const int base_index_in_ifms_tile = row_index_in_tile * parallel_w + abs_w_read;

                    if (thread_w < parallel_w / strides)
                    {
                        for (int c_h = 0; c_h < FILTER_3x3_DIM; c_h++)
                        {

                            weights_dt weight_val0 = dw_filter_weights[c_h];
                            weights_dt weight_val1 = dw_filter_weights[FILTER_3x3_DIM + c_h];
                            weights_dt weight_val2 = dw_filter_weights[FILTER_3x3_DIM * 2 + c_h];
                            weights_dt weight_val3 = dw_filter_weights[FILTER_3x3_DIM * 3 + c_h];

                            fms_dt ifms_val0 = get_fms_val(ofms_ifms_tile, row_index_in_tile + c_h, abs_w_read, dw_ifm_height, dw_ifm_width,
                                                           base_index_in_ifms_tile + c_h * parallel_w,
                                                           packed_ifm_zp);
                            fms_dt ifms_val1 = get_fms_val(ofms_ifms_tile, row_index_in_tile + c_h, abs_w_read + 1, dw_ifm_height, dw_ifm_width,
                                                           base_index_in_ifms_tile + c_h * parallel_w + 1,
                                                           packed_ifm_zp);
                            fms_dt ifms_val2 = get_fms_val(ofms_ifms_tile, row_index_in_tile + c_h, abs_w_read + 2, dw_ifm_height, dw_ifm_width,
                                                           base_index_in_ifms_tile + c_h * parallel_w + 2,
                                                           packed_ifm_zp);

                            sum0 +=
                                __dp4a(PACK_32_8s(EXTRACT_8_32(ifms_val0, 0), EXTRACT_8_32(ifms_val1, 0), EXTRACT_8_32(ifms_val2, 0), 0), weight_val0, 0);
                            sum1 +=
                                __dp4a(PACK_32_8s(EXTRACT_8_32(ifms_val0, 1), EXTRACT_8_32(ifms_val1, 1), EXTRACT_8_32(ifms_val2, 1), 0), weight_val1, 0);
                            sum2 +=
                                __dp4a(PACK_32_8s(EXTRACT_8_32(ifms_val0, 2), EXTRACT_8_32(ifms_val1, 2), EXTRACT_8_32(ifms_val2, 2), 0), weight_val2, 0);
                            sum3 +=
                                __dp4a(PACK_32_8s(EXTRACT_8_32(ifms_val0, 3), EXTRACT_8_32(ifms_val1, 3), EXTRACT_8_32(ifms_val2, 3), 0), weight_val3, 0);

                            // if (abs_row_index == 56 && abs_w_write == 5 && thread_f == 0)
                            // {
                            //     printf("%d % d %d\n", EXTRACT_8_32(ifms_val0, 0), EXTRACT_8_32(ifms_val1, 0), EXTRACT_8_32(ifms_val2, 0));
                            // }
                        }

                        // q0 = quant_relu6(sum0, scale0, fused_zp0, dw_ofms_zp, dw_relu_threshold);
                        // q1 = quant_relu6(sum1, scale1, fused_zp1, dw_ofms_zp, dw_relu_threshold);
                        // q2 = quant_relu6(sum2, scale2, fused_zp2, dw_ofms_zp, dw_relu_threshold);
                        // q3 = quant_relu6(sum3, scale3, fused_zp3, dw_ofms_zp, dw_relu_threshold);

                        // if (abs_row_index == 56 && abs_w_write == 5 && thread_f == 0)
                        // {
                        //     printf("\n%d\n", q0);
                        // }

                        // ofms_ifms_tile[row_offet_in_tile + thread_f * padded_tile_width + abs_w_write] = PACK_32_8s(q0, q1, q2, q3);
                        ofms[base_index_in_ofms] = PACK_32_8s(quant_relu6(sum0, scale0, fused_zp0, dw_ofms_zp, dw_relu_threshold),
                                                              quant_relu6(sum1, scale1, fused_zp1, dw_ofms_zp, dw_relu_threshold),
                                                              quant_relu6(sum2, scale2, fused_zp2, dw_ofms_zp, dw_relu_threshold),
                                                              quant_relu6(sum3, scale3, fused_zp3, dw_ofms_zp, dw_relu_threshold));
                    }
                }
            }
        }
    }
}

void fused_pw_dw_convolutionGPU_h_w_chw(fms_dt *ifms, fms_dt *ofms,
                                    weights_dt *pw_weights,
                                    weights_dt *dw_weights,
                                    fused_scales_dt *fused_scales,
                                    biases_dt *fused_zps,
                                    layer_specs pw_l_specs,
                                    layer_specs dw_l_specs,
                                    int *fused_params_offsets,
                                    const int iteration,
                                    float &exec_time,
                                    const int num_sms)
{

    const int num_filters = pw_l_specs.layer_num_fils;

    const int pw_ofms_width = pw_l_specs.layer_ofm_width;
    const int pw_ofms_height = pw_l_specs.layer_ofm_height;
    const int pw_compact_layer_depth = (pw_l_specs.layer_depth >> 2);

    const int dw_ofms_width = dw_l_specs.layer_ofm_width;
    const int dw_ofms_height = dw_l_specs.layer_ofm_height;
    const int dw_compact_layer_depth = (dw_l_specs.layer_depth >> 2);

    if (iteration == 0)
    {
        printf("%d, %d (FUSED_PWDW):\n", pw_l_specs.layer_index, dw_l_specs.layer_index);
    }

    const int parallel_w = TILE_W_H_W > pw_ofms_width ? least_pow_of_2_geq(pw_ofms_width) : TILE_W_H_W;
    const int parallel_h = TILE_H_H_W > pw_ofms_height  ? least_pow_of_2_geq(pw_ofms_height) : TILE_H_H_W;

    dim3 threads(parallel_w, parallel_h, 1);
    dim3 blocks((dw_ofms_width + parallel_w) / parallel_w, (dw_ofms_height + parallel_h) / parallel_h, dw_compact_layer_depth);

    uint8_t ifms_zp = (uint8_t)dw_l_specs.layer_ifms_zero_point;
    uint8_t ifm_zps_to_pack[4] = {ifms_zp, ifms_zp, ifms_zp, ifms_zp};
    fms_dt packed_ifm_zp = PACK_32_8s(ifm_zps_to_pack);

#if TIME_LAYER_BY_LAYER
    float elapsed_time;
    hipEvent_t start_event, stop_event;
    hipError_t err = hipSuccess;

    err = (hipEventCreate(&start_event));
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventCreate start_event %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = (hipEventCreate(&stop_event));
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventCreate stop_event %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipEventRecord(start_event, 0);
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventRecord start_event %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
#endif

    const int padded_tile_w = least_pow_of_2_geq(dw_l_specs.layer_ifm_width +
                                                 dw_l_specs.padding_left + dw_l_specs.padding_right);

    const int hw_parallelism = parallel_h * parallel_w;
    const int compact_layer_depth_to_parallel_hw_ratio = (pw_compact_layer_depth + hw_parallelism - 1) / hw_parallelism;

    pw_dw3x3_conv_h_w<<<blocks, threads>>>(ifms, ofms, pw_weights, fused_scales, fused_zps,
                                           pw_compact_layer_depth, num_filters,
                                           pw_l_specs.layer_ifm_width,
                                           pw_l_specs.layer_ofm_height,
                                           pw_l_specs.layer_ofm_width,
                                           pw_l_specs.layer_weights_offset / PACKED_ITEMS,
                                           fused_params_offsets[pw_l_specs.layer_index],
                                           pw_l_specs.layer_ofms_zero_point,
                                           pw_l_specs.relu_threshold,
                                           //*******************
                                           dw_weights,
                                           dw_l_specs.layer_depth,
                                           dw_l_specs.layer_ifm_height,
                                           dw_l_specs.layer_ifm_width,
                                           dw_l_specs.layer_ofm_height,
                                           dw_l_specs.layer_ofm_width,
                                           dw_l_specs.strides,
                                           dw_l_specs.padding_top,
                                           dw_l_specs.padding_bottom,
                                           dw_l_specs.padding_left,
                                           dw_l_specs.padding_right,
                                           padded_tile_w,
                                           dw_l_specs.layer_weights_offset / PACKED_ITEMS,
                                           fused_params_offsets[dw_l_specs.layer_index],
                                           dw_l_specs.layer_ifms_zero_point,
                                           dw_l_specs.layer_ofms_zero_point,
                                           packed_ifm_zp,
                                           dw_l_specs.relu_threshold,
                                           parallel_h, parallel_w,
                                           compact_layer_depth_to_parallel_hw_ratio);

#if TIME_LAYER_BY_LAYER
    err = (hipEventRecord(stop_event, 0));
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventRecord stop_event %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = (hipEventSynchronize(stop_event));
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventSynchronize %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = (hipEventElapsedTime(&elapsed_time, start_event, stop_event));
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventElapsedTime %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // printf("Measured time for sample = %.3fms\n", elapsed_time);
    if (iteration >= WARMUP_ITERATIONS)
    {
        exec_time += elapsed_time;
    }
#endif

    hipError_t kernel_error = hipGetLastError();
    if (kernel_error != hipSuccess)
    {
        cout << "the error of code: " << kernel_error << " has happened\n";
    }
}

#endif