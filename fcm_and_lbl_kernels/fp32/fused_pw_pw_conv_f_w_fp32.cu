#include "hip/hip_runtime.h"
#include "../../headers/conv_kernels.h"

#if COMPILE_FUSED && (FUSION_MODE == ALL_MODES || FUSION_MODE == FUSED_F_W) && DATA_TYPE == FLOAT_DTYPE

using namespace std;
namespace cg = cooperative_groups;

__global__ void pw_pw_conv_f_w_chw(fms_dt *ifms, fms_dt *ofms, weights_dt *pw_weights,
                               fused_scales_dt *fused_scales,
                               biases_dt *fused_zps,
                               const int pw_1_compact_layer_depth,
                               const int pw_1_compact_layer_num_filters,
                               const int pw_1_ifm_width,
                               const int pw_1_compact_ifm_width_depth,
                               const int pw_1_ofm_height,
                               const int pw_1_ofm_width,
                               const int pw_1_compact_ofm_width_depth,
                               const int pw_1_layer_weights_offset,
                               const int pw_1_layer_fused_params_offset,
                               const fms_dt pw_1_ofms_zp,
                               const scales_dt pw_1_relu_threshold,
                               const int pw_1_layer_activation,
                               const int pw_2_compact_layer_depth,
                               const int pw_2_compact_layer_num_filters,
                               const int pw_2_ifm_width,
                               const int pw_2_compact_ifm_width_depth,
                               const int pw_2_ofm_height,
                               const int pw_2_ofm_width,
                               const int pw_2_compact_ofm_width_depth,
                               const int pw_2_filters_to_parallelism_f_ratio,
                               const int pw_2_layer_weights_offset,
                               const int pw_2_layer_fused_params_offset,
                               const fms_dt pw_2_ofms_zp,
                               const scales_dt pw_2_relu_threshold,
                               const int parallel_w)
{

    const int thread_w = threadIdx.x;
    const int thread_f = threadIdx.z;

    const int block_w = blockIdx.x;
    const int abs_h_index = blockIdx.y;

    const int abs_f = thread_f * F_W_V2_TILE_F;

    const int pw_1_filter_index = abs_f * PACKED_ITEMS;

    const int pw_1_ifm_hw = pw_1_ifm_width * pw_1_ifm_width; // TODO
    const int pw_2_ofm_hw = pw_2_ofm_width * pw_2_ofm_width; // TODO

    __shared__ fms_dt ofms_ifms_tile[PW_PW_MAX_FMS_BUFFER_SZ];
    //__shared__ fms_dt pw_2_weights[PW_PW_MAX_WEIGHTS_BUFFER_SZ];

    scales_dt scale0, scale1, scale2, scale3;

    biases_dt fused_zp0, fused_zp1, fused_zp2, fused_zp3;

    const int abs_w_index = block_w * parallel_w + thread_w;

    const int base_index_pw_1_weights = pw_1_layer_weights_offset + pw_1_filter_index * pw_1_compact_layer_depth;
    const int base_index_ifms = abs_h_index * pw_1_ifm_width + abs_w_index;

    if (abs_w_index < pw_1_ofm_width)
    {
        pss_dt sum0 = 0, sum1 = 0, sum2 = 0, sum3 = 0;
        for (int d = 0; d < pw_1_compact_layer_depth; d++)
        {
            fms_dt fms_val = ifms[base_index_ifms + d * pw_1_ifm_hw];
            // ifms_tile[d * parallel_w + thread_w];

            sum0 += fms_val * pw_weights[base_index_pw_1_weights + d];
            sum1 += fms_val * pw_weights[base_index_pw_1_weights + pw_1_compact_layer_depth + d];
            sum2 += fms_val * pw_weights[base_index_pw_1_weights + 2 * pw_1_compact_layer_depth + d];
            sum3 += fms_val * pw_weights[base_index_pw_1_weights + 3 * pw_1_compact_layer_depth + d];
        }
        if (sum0 < 0)
        {
            sum0 = 0;
        }
        else
        {
            sum0 = sum0 * DUMMY_SCALE + DUMMY_BIAS;
        }
        if (sum1 < 0)
        {
            sum1 = 0;
        }
        else
        {
            sum1 = sum1 * DUMMY_SCALE + DUMMY_BIAS;
        }
        if (sum2 < 0)
        {
            sum2 = 0;
        }
        else
        {
            sum2 = sum2 * DUMMY_SCALE + DUMMY_BIAS;
        }
        if (sum3 < 0)
        {
            sum3 = 0;
        }
        else
        {
            sum3 = sum3 * DUMMY_SCALE + DUMMY_BIAS;
        }
        ofms_ifms_tile[abs_f * parallel_w + thread_w] = sum0;
        ofms_ifms_tile[(abs_f + 1) * parallel_w + thread_w] = sum1;
        ofms_ifms_tile[(abs_f + 2) * parallel_w + thread_w] = sum2;
        ofms_ifms_tile[(abs_f + 3) * parallel_w + thread_w] = sum3;
    }
    //}

    __syncthreads();
    // if (thread_w >= parallel_w)
    {
        // const int abs_w_index = block_w * parallel_w + (thread_w - parallel_w);

        for (int o_f = 0; o_f < pw_2_filters_to_parallelism_f_ratio; o_f++)
        {
            const int pw_2_compact_filter_index = abs_f * pw_2_filters_to_parallelism_f_ratio + o_f * F_W_V2_TILE_F;
            const int pw_2_filter_index = pw_2_compact_filter_index * PACKED_ITEMS;

            if (pw_2_compact_filter_index < pw_2_compact_layer_num_filters)
            {
                const int f_offset = pw_2_filter_index * pw_2_compact_layer_depth;
                const int base_index_pw_2_weights = pw_2_layer_weights_offset + f_offset;

                if (abs_w_index < pw_2_ofm_width)
                {
                    const int base_index_in_ofms = abs_h_index * pw_2_ofm_width +
                                                   pw_2_compact_filter_index * pw_2_ofm_hw + abs_w_index;

                    pss_dt sum0 = 0, sum1 = 0, sum2 = 0, sum3 = 0;
                    int a = 0;
                    for (int d = 0; d < pw_2_compact_layer_depth; d++)
                    {
                        fms_dt fms_val = ofms_ifms_tile[d * parallel_w + thread_w];

                        sum0 += fms_val * pw_weights[base_index_pw_2_weights + d];
                        sum1 += fms_val * pw_weights[base_index_pw_2_weights + pw_2_compact_layer_depth + d];
                        sum2 += fms_val * pw_weights[base_index_pw_2_weights + 2 * pw_2_compact_layer_depth + d];
                        sum3 += fms_val * pw_weights[base_index_pw_2_weights + 3 * pw_2_compact_layer_depth + d];
                    }
                    if (sum0 < 0)
                    {
                        sum0 = 0;
                    }
                    else
                    {
                        sum0 = sum0 * DUMMY_SCALE + DUMMY_BIAS;
                    }
                    if (sum1 < 0)
                    {
                        sum1 = 0;
                    }
                    else
                    {
                        sum1 = sum1 * DUMMY_SCALE + DUMMY_BIAS;
                    }
                    if (sum2 < 0)
                    {
                        sum2 = 0;
                    }
                    else
                    {
                        sum2 = sum2 * DUMMY_SCALE + DUMMY_BIAS;
                    }
                    if (sum3 < 0)
                    {
                        sum3 = 0;
                    }
                    else
                    {
                        sum3 = sum3 * DUMMY_SCALE + DUMMY_BIAS;
                    }

                    ofms[base_index_in_ofms] = sum0;
                    ofms[base_index_in_ofms + pw_2_ofm_hw] = sum1;
                    ofms[base_index_in_ofms + 2 * pw_2_ofm_hw] = sum2;
                    ofms[base_index_in_ofms + 3 * pw_2_ofm_hw] = sum3;
                }
            }
        }
    }
}

void fused_pw_pw_convolutionGPU_chw(fms_dt *ifms, fms_dt *ofms,
                                weights_dt *pw_weights,
                                fused_scales_dt *fused_scales,
                                biases_dt *fused_zps,
                                layer_specs pw_1_l_specs,
                                layer_specs pw_2_l_specs,
                                int *fused_params_offsets,
                                const int iteration,
                                int *layers_parallelism_w,
                                float &exec_time)
{

    const int pw_1_ofms_width = pw_1_l_specs.layer_ofm_width;
    const int pw_1_ofms_height = pw_1_l_specs.layer_ofm_height;
    const int pw_1_compact_layer_depth = (pw_1_l_specs.layer_depth / PACKED_ITEMS);
    const int pw_1_compact_layer_num_filters = (pw_1_l_specs.layer_num_fils / PACKED_ITEMS);

    int parallel_w = layers_parallelism_w[pw_1_l_specs.layer_index];

    dim3 threads(parallel_w, 1, pw_1_compact_layer_num_filters / F_W_V2_TILE_F);
    dim3 blocks((pw_1_l_specs.layer_ifm_width + parallel_w - 1) / parallel_w,
                pw_1_l_specs.layer_ifm_height, 1);

#if TIME_LAYER_BY_LAYER
    float elapsed_time;
    hipEvent_t start_event, stop_event;
    hipError_t err = hipSuccess;

    err = (hipEventCreate(&start_event));
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventCreate start_event %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = (hipEventCreate(&stop_event));
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventCreate stop_event %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    if (iteration == 0)
    {
        printf("%d, %d (FUSED_PWPW):\n", pw_1_l_specs.layer_index, pw_2_l_specs.layer_index);
    }

    err = hipEventRecord(start_event, 0);
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventRecord start_event %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
#endif

    const int pw_1_layer_depth = pw_1_l_specs.layer_depth;
    const int pw_1_layer_ifm_width = pw_1_l_specs.layer_ifm_width;
    const int pw_1_layer_ofm_width = pw_1_l_specs.layer_ofm_width;
    const int pw_1_compact_ofms_width_depth = pw_1_compact_layer_num_filters * pw_1_layer_ofm_width;
    const int pw_2_layer_depth = pw_2_l_specs.layer_depth;
    const int pw_2_layer_ifm_width = pw_2_l_specs.layer_ifm_width;
    const int pw_2_layer_ofm_width = pw_2_l_specs.layer_ofm_width;
    const int pw_2_compact_layer_num_filters = pw_2_l_specs.layer_num_fils / PACKED_ITEMS;
    const int pw_2_compact_ofms_width_depth = pw_2_compact_layer_num_filters * pw_2_layer_ofm_width;
    const int pw_2_compact_layer_depth = pw_2_layer_depth / PACKED_ITEMS;

    int pw_2_filters_to_parallelism_f_ratio = pw_2_compact_layer_num_filters / (pw_1_compact_layer_num_filters / F_W_V2_TILE_F);
    if (pw_2_filters_to_parallelism_f_ratio < 1)
    {
        pw_2_filters_to_parallelism_f_ratio = 1;
    }

    pw_pw_conv_f_w_chw<<<blocks, threads>>>(ifms, ofms, pw_weights,
                                        fused_scales,
                                        fused_zps,
                                        pw_1_compact_layer_depth,
                                        pw_1_compact_layer_num_filters,
                                        pw_1_layer_ifm_width,
                                        pw_1_compact_layer_depth * pw_1_layer_ifm_width,
                                        pw_1_l_specs.layer_ofm_height,
                                        pw_1_l_specs.layer_ofm_width,
                                        pw_1_compact_ofms_width_depth,
                                        pw_1_l_specs.layer_weights_offset / PACKED_ITEMS,
                                        fused_params_offsets[pw_1_l_specs.layer_index],
                                        pw_1_l_specs.layer_ofms_zero_point,
                                        pw_1_l_specs.relu_threshold,
                                        pw_1_l_specs.layer_activation,
                                        pw_2_compact_layer_depth,
                                        pw_2_compact_layer_num_filters,
                                        pw_2_layer_ifm_width,
                                        pw_2_compact_layer_depth * pw_2_layer_ifm_width,
                                        pw_2_l_specs.layer_ofm_height,
                                        pw_2_l_specs.layer_ofm_width,
                                        pw_2_compact_ofms_width_depth,
                                        pw_2_filters_to_parallelism_f_ratio,
                                        pw_2_l_specs.layer_weights_offset / PACKED_ITEMS,
                                        fused_params_offsets[pw_2_l_specs.layer_index],
                                        pw_2_l_specs.layer_ofms_zero_point,
                                        pw_2_l_specs.relu_threshold,
                                        parallel_w);

#if TIME_LAYER_BY_LAYER
    err = (hipEventRecord(stop_event, 0));
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventRecord stop_event %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = (hipEventSynchronize(stop_event));
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventSynchronize %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = (hipEventElapsedTime(&elapsed_time, start_event, stop_event));
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventElapsedTime %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // printf("Measured time for sample = %.3fms\n", elapsed_time);
    if (iteration >= WARMUP_ITERATIONS)
    {
        exec_time += elapsed_time;
    }
#endif

    hipError_t kernel_error = hipGetLastError();
    if (kernel_error != hipSuccess)
    {
        cout << "the error of code: " << kernel_error << " has happened\n";
    }
}

#endif