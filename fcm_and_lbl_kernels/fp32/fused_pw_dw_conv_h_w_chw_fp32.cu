#include "hip/hip_runtime.h"
#include "../../headers/conv_kernels.h"

#if COMPILE_FUSED && (FUSION_MODE == ALL_MODES || FUSION_MODE == NOT_FUSED) && DATA_TYPE == FLOAT_DTYPE

__global__ void pw_dw3x3_conv_h_w_chw(fms_dt *ifms, fms_dt *ofms, weights_dt *pw_weights,
                                  fused_scales_dt *fused_scales,
                                  biases_dt *fused_zps,
                                  const int compact_layer_depth,
                                  const int pw_num_filters,
                                  const int pw_ifm_width,
                                  const int pw_ofm_height,
                                  const int pw_ofm_width,
                                  const int pw_layer_weights_offset,
                                  const int pw_layer_fused_params_offset,
                                  const fms_dt pw_ofms_zp,
                                  const scales_dt pw_relu_threshold,
                                  weights_dt *dw_weights,
                                  const int dw_ifm_depth,
                                  const int dw_ifm_height,
                                  const int dw_ifm_width,
                                  const int dw_ofm_height,
                                  const int dw_ofm_width,
                                  const int strides,
                                  const int padding_top,
                                  const int padding_bottom,
                                  const int padding_left,
                                  const int padding_right,
                                  const int padded_tile_width,
                                  const int dw_layer_weights_offset,
                                  const int dw_layer_fused_params_offset,
                                  const fms_dt dw_ifms_zp,
                                  const fms_dt dw_ofms_zp,
                                  const scales_dt dw_relu_threshold,
                                  const int parallel_h,
                                  const int parallel_w,
                                  const int compact_layer_depth_to_parallel_hw_ratio)
{

    const int thread_w = threadIdx.x;
    const int thread_h = threadIdx.y;
    const int thread_f = threadIdx.z;

    const int block_w = blockIdx.x;
    const int block_h = blockIdx.y;
    const int block_f = blockIdx.z;

    const int dw_ofms_hw = dw_ofm_height * dw_ofm_width;

    const int parallel_hw = parallel_h * parallel_w;

    __shared__ weights_dt dw_filter_weights[TILE_F_H_W][FILTER_3x3_PADDED_AREA];
    __shared__ weights_dt pw_weights_tile[TILE_F_H_W][256];
    __shared__ fms_dt ofms_ifms_tile[TILE_F_H_W * TILE_H_H_W * TILE_W_H_W];

    const int abs_f_compact = block_f * TILE_F_H_W + thread_f;
    int base_index_pw_weights = pw_layer_weights_offset + abs_f_compact * PACKED_ITEMS * compact_layer_depth;
    const int thread_hw = thread_h * parallel_h + thread_w;

    for (int o_f = 0; o_f < TILE_F_H_W; o_f++)
    {
        if (thread_h == 0 && thread_w < FILTER_3x3_DIM)
        {
            for (int c_h = 0; c_h < FILTER_3x3_DIM; c_h++)
            {
                dw_filter_weights[o_f][c_h * FILTER_3x3_DIM + thread_w] =
                    dw_weights[dw_layer_weights_offset + (c_h * FILTER_3x3_DIM + thread_w) +
                               (abs_f_compact + o_f) * FILTER_3x3_PADDED_AREA];
            }
        }
    }
    // ****************************************************************************************************
    pss_dt sum0 = 0, sum1 = 0, sum2 = 0, sum3 = 0;
#if TILE_F_H_W == 8
    pss_dt sum4 = 0, sum5 = 0, sum6 = 0, sum7 = 0;
#endif

    const int abs_w_write = block_w * parallel_w + thread_w;
    const int abs_w_read = block_w * parallel_w + thread_w; // TODO
    const int offet_in_tile_hw = thread_h * parallel_w + thread_w;

    for (int i = 0; i < compact_layer_depth_to_parallel_hw_ratio; i++)
    {
        const int iter_ifms_offset = i * parallel_hw;
        for (int o_f = 0; o_f < TILE_F_H_W; o_f++)
        {
            if (iter_ifms_offset + thread_hw < compact_layer_depth) // TODO
            {
                pw_weights_tile[o_f][thread_hw] = pw_weights[base_index_pw_weights + iter_ifms_offset +
                                                             (o_f * compact_layer_depth) + thread_hw];
            }
        }

        __syncthreads();

        if ((block_h * parallel_h + thread_h) < pw_ofm_width && abs_w_write < pw_ofm_width)
        {

            for (int d = 0; d < parallel_hw && d + iter_ifms_offset < compact_layer_depth; d++)
            {
                int base_index_in_ifms = (block_h * parallel_h + thread_h) * pw_ifm_width +
                                         (d + iter_ifms_offset) * pw_ifm_width * pw_ifm_width + abs_w_read; // TODO

                fms_dt ifms_val = ifms[base_index_in_ifms];
                sum0 += ifms_val * pw_weights_tile[0][d];
                sum1 += ifms_val * pw_weights_tile[1][d];
                sum2 += ifms_val * pw_weights_tile[2][d];
                sum3 += ifms_val * pw_weights_tile[3][d];
#if TILE_F_H_W == 8
                sum4 += ifms_val * pw_weights_tile[4][d];
                sum5 += ifms_val * pw_weights_tile[5][d];
                sum6 += ifms_val * pw_weights_tile[6][d];
                sum7 += ifms_val * pw_weights_tile[7][d];
#endif
            }
        }
        __syncthreads();
    }
    if ((block_h * parallel_h + thread_h) < pw_ofm_width && abs_w_write < pw_ofm_width)
    {
        if (sum0 < 0)
        {
            sum0 = 0;
        }
        else
        {
            sum0 = sum0 * DUMMY_SCALE + DUMMY_BIAS;
        }
        if (sum1 < 0)
        {
            sum1 = 0;
        }
        else
        {
            sum1 = sum1 * DUMMY_SCALE + DUMMY_BIAS;
        }
        if (sum2 < 0)
        {
            sum2 = 0;
        }
        else
        {
            sum2 = sum2 * DUMMY_SCALE + DUMMY_BIAS;
        }
        if (sum3 < 0)
        {
            sum3 = 0;
        }
        else
        {
            sum3 = sum3 * DUMMY_SCALE + DUMMY_BIAS;
        }
#if TILE_F_H_W == 8
        if (sum4 < 0)
        {
            sum4 = 0;
        }
        else
        {
            sum4 = sum4 * DUMMY_SCALE + DUMMY_BIAS;
        }
        if (sum5 < 0)
        {
            sum5 = 0;
        }
        else
        {
            sum5 = sum5 * DUMMY_SCALE + DUMMY_BIAS;
        }
        if (sum6 < 0)
        {
            sum6 = 0;
        }
        else
        {
            sum6 = sum6 * DUMMY_SCALE + DUMMY_BIAS;
        }
        if (sum7 < 0)
        {
            sum7 = 0;
        }
        else
        {
            sum7 = sum7 * DUMMY_SCALE + DUMMY_BIAS;
        }
#endif

        ofms_ifms_tile[offet_in_tile_hw] = sum0;
        ofms_ifms_tile[TILE_HW_H_W + offet_in_tile_hw] = sum1;
        ofms_ifms_tile[2 * TILE_HW_H_W + offet_in_tile_hw] = sum2;
        ofms_ifms_tile[3 * TILE_HW_H_W + offet_in_tile_hw] = sum3;
#if TILE_F_H_W == 8
        ofms_ifms_tile[4 * TILE_HW_H_W + offet_in_tile_hw] = sum4;
        ofms_ifms_tile[5 * TILE_HW_H_W + offet_in_tile_hw] = sum5;
        ofms_ifms_tile[6 * TILE_HW_H_W + offet_in_tile_hw] = sum6;
        ofms_ifms_tile[7 * TILE_HW_H_W + offet_in_tile_hw] = sum7;
#endif
    }

    __syncthreads();
    // ****************************************************************************************************
    {
        const int abs_row_index = (block_h * parallel_h + thread_h) * strides - padding_top;

        if ((block_h * parallel_h + thread_h) < dw_ofm_height)
        {
            const int abs_w_write = block_w * parallel_w + thread_w;
            const int w_read = thread_w * strides - padding_left;

            int base_index_in_ofms = abs_f_compact * dw_ofms_hw +
                                     (block_h * parallel_h + thread_h) * dw_ofm_width + abs_w_write;

            if (abs_w_write < dw_ofm_width)
            {
                const int row_index_in_tile = thread_h * strides - padding_top;
                int base_index_in_ifms_tile = row_index_in_tile * parallel_w + w_read;

                sum0 = 0, sum1 = 0, sum2 = 0, sum3 = 0;
#if TILE_F_H_W == 8
                sum4 = 0, sum5 = 0, sum6 = 0, sum7 = 0;
#endif
                for (int c_h = 0; c_h < FILTER_3x3_DIM; c_h++)
                {
                    for (int c_w = 0; c_w < FILTER_3x3_DIM; c_w++)
                    {

                        fms_dt ifms_val0, ifms_val1, ifms_val2, ifms_val3;
#if TILE_F_H_W == 8
                        fms_dt ifms_val4, ifms_val5, ifms_val6, ifms_val7;
#endif
                        get_fms_vals(ofms_ifms_tile, row_index_in_tile + c_h,
                                     w_read + c_w, dw_ifm_height, dw_ifm_width,
                                     base_index_in_ifms_tile + c_h * parallel_w + c_w,
                                     dw_ifms_zp,
                                     TILE_HW_H_W,
                                     ifms_val0,
                                     ifms_val1,
                                     ifms_val2,
                                     ifms_val3);
#if TILE_F_H_W == 8
                        get_fms_vals(ofms_ifms_tile, row_index_in_tile + c_h,
                                     w_read + c_w, dw_ifm_height, dw_ifm_width,
                                     base_index_in_ifms_tile + 4 * TILE_HW_H_W + c_h * parallel_w + c_w,
                                     dw_ifms_zp,
                                     TILE_HW_H_W,
                                     ifms_val4,
                                     ifms_val5,
                                     ifms_val6,
                                     ifms_val7);
#endif

                        sum0 += dw_filter_weights[0][c_h * FILTER_3x3_DIM + c_w] * ifms_val0;
                        sum1 += dw_filter_weights[1][c_h * FILTER_3x3_DIM + c_w] * ifms_val1;
                        sum2 += dw_filter_weights[2][c_h * FILTER_3x3_DIM + c_w] * ifms_val2;
                        sum3 += dw_filter_weights[3][c_h * FILTER_3x3_DIM + c_w] * ifms_val3;
#if TILE_F_H_W == 8
                        sum4 += dw_filter_weights[4][c_h * FILTER_3x3_DIM + c_w] * ifms_val4;
                        sum5 += dw_filter_weights[5][c_h * FILTER_3x3_DIM + c_w] * ifms_val5;
                        sum6 += dw_filter_weights[6][c_h * FILTER_3x3_DIM + c_w] * ifms_val6;
                        sum7 += dw_filter_weights[7][c_h * FILTER_3x3_DIM + c_w] * ifms_val7;
#endif
                    }
                }
                if (sum0 < 0)
                {
                    sum0 = 0;
                }
                else
                {
                    sum0 = sum0 * DUMMY_SCALE + DUMMY_BIAS;
                }
                if (sum1 < 0)
                {
                    sum1 = 0;
                }
                else
                {
                    sum1 = sum1 * DUMMY_SCALE + DUMMY_BIAS;
                }
                if (sum2 < 0)
                {
                    sum2 = 0;
                }
                else
                {
                    sum2 = sum2 * DUMMY_SCALE + DUMMY_BIAS;
                }
                if (sum3 < 0)
                {
                    sum3 = 0;
                }
                else
                {
                    sum3 = sum3 * DUMMY_SCALE + DUMMY_BIAS;
                }
#if TILE_F_H_W == 8
                if (sum4 < 0)
                {
                    sum4 = 0;
                }
                else
                {
                    sum4 = sum4 * DUMMY_SCALE + DUMMY_BIAS;
                }
                if (sum5 < 0)
                {
                    sum5 = 0;
                }
                else
                {
                    sum5 = sum5 * DUMMY_SCALE + DUMMY_BIAS;
                }
                if (sum6 < 0)
                {
                    sum6 = 0;
                }
                else
                {
                    sum6 = sum6 * DUMMY_SCALE + DUMMY_BIAS;
                }
                if (sum7 < 0)
                {
                    sum7 = 0;
                }
                else
                {
                    sum7 = sum7 * DUMMY_SCALE + DUMMY_BIAS;
                }
#endif
                ofms[base_index_in_ofms] = sum0;
                ofms[base_index_in_ofms + dw_ofms_hw] = sum1;
                ofms[base_index_in_ofms + 2 * dw_ofms_hw] = sum2;
                ofms[base_index_in_ofms + 3 * dw_ofms_hw] = sum3;
#if TILE_F_H_W == 8
                ofms[base_index_in_ofms + 4 * dw_ofms_hw] = sum4;
                ofms[base_index_in_ofms + 5 * dw_ofms_hw] = sum5;
                ofms[base_index_in_ofms + 6 * dw_ofms_hw] = sum6;
                ofms[base_index_in_ofms + 7 * dw_ofms_hw] = sum7;
#endif
                //}
            }
        }
    }
}

void fused_pw_dw_convolutionGPU_h_w_chw(fms_dt *ifms, fms_dt *ofms,
                                    weights_dt *pw_weights,
                                    weights_dt *dw_weights,
                                    fused_scales_dt *fused_scales,
                                    biases_dt *fused_zps,
                                    layer_specs pw_l_specs,
                                    layer_specs dw_l_specs,
                                    int *fused_params_offsets,
                                    const int iteration,
                                    float &exec_time,
                                    const int num_sms)
{

    const int num_filters = pw_l_specs.layer_num_fils;

    const int pw_ofms_width = pw_l_specs.layer_ofm_width;
    const int pw_ofms_height = pw_l_specs.layer_ofm_height;
    const int pw_compact_layer_depth = pw_l_specs.layer_depth / PACKED_ITEMS;

    const int dw_ofms_width = dw_l_specs.layer_ofm_width;
    const int dw_ofms_height = dw_l_specs.layer_ofm_height;
    const int dw_compact_layer_depth = dw_l_specs.layer_depth / PACKED_ITEMS;

    if (iteration == 0)
    {
        printf("%d, %d (FUSED_PWDW):\n", pw_l_specs.layer_index, dw_l_specs.layer_index);
    }

    const int compact_layer_depth = dw_l_specs.layer_depth / PACKED_ITEMS;

    const int parallel_w = TILE_W_H_W > pw_ofms_width ? least_pow_of_2_geq(pw_ofms_width) : TILE_W_H_W;
    const int parallel_h = TILE_H_H_W > pw_ofms_height  ? least_pow_of_2_geq(pw_ofms_height) : TILE_H_H_W;

    dim3 threads(parallel_w, parallel_h, 1);
    dim3 blocks((dw_ofms_width + parallel_w - 1) / parallel_w,
                (dw_ofms_height + parallel_h - 1) / parallel_h, dw_compact_layer_depth / TILE_F_H_W);

    uint8_t ifms_zp = (uint8_t)dw_l_specs.layer_ifms_zero_point;

#if TIME_LAYER_BY_LAYER
    float elapsed_time;
    hipEvent_t start_event, stop_event;
    hipError_t err = hipSuccess;

    err = (hipEventCreate(&start_event));
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventCreate start_event %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = (hipEventCreate(&stop_event));
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventCreate stop_event %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipEventRecord(start_event, 0);
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventRecord start_event %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
#endif

    const int padded_tile_w = least_pow_of_2_geq(dw_l_specs.layer_ifm_width +
                                                 dw_l_specs.padding_left + dw_l_specs.padding_right);

    const int parallelism_hw = parallel_h * parallel_w;
    const int compact_layer_depth_to_parallel_hw_ratio = (pw_compact_layer_depth + parallelism_hw - 1) / parallelism_hw;

    pw_dw3x3_conv_h_w_chw<<<blocks, threads>>>(ifms, ofms, pw_weights, fused_scales, fused_zps,
                                           pw_compact_layer_depth, num_filters,
                                           pw_l_specs.layer_ifm_width,
                                           pw_l_specs.layer_ofm_height,
                                           pw_l_specs.layer_ofm_width,
                                           pw_l_specs.layer_weights_offset / PACKED_ITEMS,
                                           fused_params_offsets[pw_l_specs.layer_index],
                                           pw_l_specs.layer_ofms_zero_point,
                                           pw_l_specs.relu_threshold,
                                           //*******************
                                           dw_weights,
                                           dw_l_specs.layer_depth,
                                           dw_l_specs.layer_ifm_height,
                                           dw_l_specs.layer_ifm_width,
                                           dw_l_specs.layer_ofm_height,
                                           dw_l_specs.layer_ofm_width,
                                           dw_l_specs.strides,
                                           dw_l_specs.padding_top,
                                           dw_l_specs.padding_bottom,
                                           dw_l_specs.padding_left,
                                           dw_l_specs.padding_right,
                                           padded_tile_w,
                                           dw_l_specs.layer_weights_offset / PACKED_ITEMS,
                                           fused_params_offsets[dw_l_specs.layer_index],
                                           dw_l_specs.layer_ifms_zero_point,
                                           dw_l_specs.layer_ofms_zero_point,
                                           dw_l_specs.relu_threshold,
                                           parallel_h, parallel_w,
                                           compact_layer_depth_to_parallel_hw_ratio);

#if TIME_LAYER_BY_LAYER
    err = (hipEventRecord(stop_event, 0));
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventRecord stop_event %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = (hipEventSynchronize(stop_event));
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventSynchronize %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = (hipEventElapsedTime(&elapsed_time, start_event, stop_event));
    if (err != hipSuccess)
    {
        fprintf(stderr,
                "Failed to hipEventElapsedTime %s\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // printf("Measured time for sample = %.3fms\n", elapsed_time);
    if (iteration >= WARMUP_ITERATIONS)
    {
        exec_time += elapsed_time;
    }
#endif

    hipError_t kernel_error = hipGetLastError();
    if (kernel_error != hipSuccess)
    {
        cout << "the error of code: " << kernel_error << " has happened\n";
    }
}

#endif